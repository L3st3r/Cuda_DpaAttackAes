#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include <string>
using namespace std;

/*
*  Global Configuration
*/
#define GPU             // {CPU, GPU}
#define NAIVE           // {NAIVE, SHARED}  SHARED funktioniert noch nicht
#define HW_SL           // {HW_SL, HW_PL}
#define HW_PL

#define NUMBER_OF_TRACES 10000
#define POINTS_PER_TRACE 10000
#define NUMBER_OF_TEXTS 10000
#define BYTES_PER_TEXT 16     // fix for AES
#define BYTES_PER_KEY 1       // possible values for AES are 16, 24 and 32 (128, 192 or 256 bits)
#define NUMBER_OF_KEY_CANDIDATES 256

#define TRACE_STARTPOINT 550
#define TRACE_ENDPOINT 600

const int NUMBER_OF_TRACES_INT = 10000;

string TRACE_FILE = "Traces00000.dat";
string PLAINTEXT_FILE = "plaintexts.dat";
string CIPHERTEXT_FILE = "ciphertexts.dat";

/*

CORRECT CIPHER KEY: 2b  7e  15  16  28  ae  d2  a6  ab  f7  15  88  09  cf  4f  3c
dec: 043 126 021 022 040 174 210 166 171 247 021 136 009 207 079 060		 
*/

hipError_t getHwWithCuda(int *hw, int *plaintext);

// #################### CURRENTLY UNUSED FUNCTIONS #####################



void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %u\n",  devProp.totalConstMem);
    printf("Texture alignment:             %u\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

void printDevInfos()
{
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }

    printf("\nPress any key to exit...");
    char c;
    scanf("%c", &c);
}

//-- dynamic 2d arrays with contiguously stored data
// copied from https://devtalk.nvidia.com/
template <class T> class array2d
{
    T *data;
    size_t R, C;

public:
    array2d (size_t row, size_t col)
    {
        data = new T [row*col];
        R = row;
        C = col;
    }

    T* operator[] (size_t row) { return &(data[row*C]);	}
    T* operator& () { return &(data[0]); }
    ~array2d () { delete [] data; }
};

/*
*	Function to read in values of traces
*/
void read_traces_array(array2d<int> traces_array, string filename) {
    streampos size;
    char * memblock;

    ifstream file (filename, ios::in|ios::binary|ios::ate);
    if (file.is_open())
    {
        size = file.tellg();
        memblock = new char [size];
        file.seekg (0, ios::beg);
        file.read (memblock, size);
        file.close();

        cout << "Content of file " << filename << " is in memory." << endl;   
    }
    else
    {
        cout << "Unable to open file" << filename << endl;
        return;
    }

    for (int i = 0; i < NUMBER_OF_TRACES; i++)
    {
        for (int j = 0; j < POINTS_PER_TRACE; j++)
        {
            //traces[i][j] = static_cast<int>(memblock[i*POINTS_PER_TRACE + j]);
            traces_array[j][i] = static_cast<int>(memblock[i*POINTS_PER_TRACE + j]);   // easier this way, so we don't need the array traces_at_tracepoint
            //cout << static_cast<int>(memblock[i]);
        }
    }
    delete[] memblock;
}

#

// #################### ACTUAL PROGRAM #####################

/*
*	Function to read in values of traces
*/
void read_traces(int *traces, string filename) {
    streampos size;
    char * memblock;

    ifstream file (filename, ios::in|ios::binary|ios::ate);
    if (file.is_open())
    {
        size = file.tellg();
        memblock = new char [size];
        file.seekg (0, ios::beg);
        file.read (memblock, size);
        file.close();

        cout << "Content of file " << filename << " is in memory." << endl;   
    }
    else
    {
        cout << "Unable to open file" << filename << endl;
        return;
    }

    for (int i = 0; i < NUMBER_OF_TRACES*POINTS_PER_TRACE; i++)
    {
        traces[i] = static_cast<int>(memblock[i]);   // easier this way, so we don't need the array traces_at_tracepoint
    }

    //for (int i = 0; i < NUMBER_OF_TRACES; i++)
    //{
    // for (int j = 0; j < POINTS_PER_TRACE; j++)
    // {
    //   //traces[i][j] = static_cast<int>(memblock[i*POINTS_PER_TRACE + j]);
    //   traces[j][i] = static_cast<int>(memblock[i*POINTS_PER_TRACE + j]);   // easier this way, so we don't need the array traces_at_tracepoint
    //   //cout << static_cast<int>(memblock[i]);
    // }
    //}
    delete[] memblock;
}


/*
*	Function to read in plaintexts or ciphertexts
*/
void read_texts(unsigned _int8 **texts, string filename) {
    streampos size;
    char * memblock;

    ifstream file (filename, ios::in|ios::binary|ios::ate);
    if (file.is_open())
    {
        size = file.tellg();
        memblock = new char [size];
        file.seekg (0, ios::beg);
        file.read (memblock, size);
        file.close();

        cout << "Content of file " << filename << " is in memory." << endl;    
    }
    else
    {
        cout << "Unable to open file " << filename << endl;
        return;
    }

    for (int j = 0; j < BYTES_PER_TEXT; j++)
    {
        for (int i = 0; i < NUMBER_OF_TEXTS; i++)
        {
            texts[i][j] = memblock[j*NUMBER_OF_TEXTS + i];
        }
    }
    delete[] memblock;
}


/*
*	Function that computes the hamming weight
*/
unsigned int get_Hw(unsigned int b)
{
    unsigned int hw = 0;
    while (b) 
    {
        hw += (b & 1);
        b >>= 1;
    }
    return hw;
}


/*
*	Function that computes the T-Table output for a plaintext-byte and a key candidate
*/
unsigned int get_TTable_Out(unsigned int plaintext_byte, unsigned int key_candidate)
{
    // AES T-Table LUT
    unsigned int ttable0 [] = 
    { 0xC66363A5, 0xF87C7C84, 0xEE777799, 0xF67B7B8D, 0xFFF2F20D, 0xD66B6BBD, 0xDE6F6FB1, 0x91C5C554, 
    0x60303050, 0x02010103, 0xCE6767A9, 0x562B2B7D, 0xE7FEFE19, 0xB5D7D762, 0x4DABABE6, 0xEC76769A, 
    0x8FCACA45, 0x1F82829D, 0x89C9C940, 0xFA7D7D87, 0xEFFAFA15, 0xB25959EB, 0x8E4747C9, 0xFBF0F00B, 
    0x41ADADEC, 0xB3D4D467, 0x5FA2A2FD, 0x45AFAFEA, 0x239C9CBF, 0x53A4A4F7, 0xE4727296, 0x9BC0C05B, 
    0x75B7B7C2, 0xE1FDFD1C, 0x3D9393AE, 0x4C26266A, 0x6C36365A, 0x7E3F3F41, 0xF5F7F702, 0x83CCCC4F, 
    0x6834345C, 0x51A5A5F4, 0xD1E5E534, 0xF9F1F108, 0xE2717193, 0xABD8D873, 0x62313153, 0x2A15153F, 
    0x0804040C, 0x95C7C752, 0x46232365, 0x9DC3C35E, 0x30181828, 0x379696A1, 0x0A05050F, 0x2F9A9AB5, 
    0x0E070709, 0x24121236, 0x1B80809B, 0xDFE2E23D, 0xCDEBEB26, 0x4E272769, 0x7FB2B2CD, 0xEA75759F, 
    0x1209091B, 0x1D83839E, 0x582C2C74, 0x341A1A2E, 0x361B1B2D, 0xDC6E6EB2, 0xB45A5AEE, 0x5BA0A0FB, 
    0xA45252F6, 0x763B3B4D, 0xB7D6D661, 0x7DB3B3CE, 0x5229297B, 0xDDE3E33E, 0x5E2F2F71, 0x13848497, 
    0xA65353F5, 0xB9D1D168, 0x00000000, 0xC1EDED2C, 0x40202060, 0xE3FCFC1F, 0x79B1B1C8, 0xB65B5BED, 
    0xD46A6ABE, 0x8DCBCB46, 0x67BEBED9, 0x7239394B, 0x944A4ADE, 0x984C4CD4, 0xB05858E8, 0x85CFCF4A, 
    0xBBD0D06B, 0xC5EFEF2A, 0x4FAAAAE5, 0xEDFBFB16, 0x864343C5, 0x9A4D4DD7, 0x66333355, 0x11858594, 
    0x8A4545CF, 0xE9F9F910, 0x04020206, 0xFE7F7F81, 0xA05050F0, 0x783C3C44, 0x259F9FBA, 0x4BA8A8E3, 
    0xA25151F3, 0x5DA3A3FE, 0x804040C0, 0x058F8F8A, 0x3F9292AD, 0x219D9DBC, 0x70383848, 0xF1F5F504, 
    0x63BCBCDF, 0x77B6B6C1, 0xAFDADA75, 0x42212163, 0x20101030, 0xE5FFFF1A, 0xFDF3F30E, 0xBFD2D26D, 
    0x81CDCD4C, 0x180C0C14, 0x26131335, 0xC3ECEC2F, 0xBE5F5FE1, 0x359797A2, 0x884444CC, 0x2E171739, 
    0x93C4C457, 0x55A7A7F2, 0xFC7E7E82, 0x7A3D3D47, 0xC86464AC, 0xBA5D5DE7, 0x3219192B, 0xE6737395, 
    0xC06060A0, 0x19818198, 0x9E4F4FD1, 0xA3DCDC7F, 0x44222266, 0x542A2A7E, 0x3B9090AB, 0x0B888883, 
    0x8C4646CA, 0xC7EEEE29, 0x6BB8B8D3, 0x2814143C, 0xA7DEDE79, 0xBC5E5EE2, 0x160B0B1D, 0xADDBDB76, 
    0xDBE0E03B, 0x64323256, 0x743A3A4E, 0x140A0A1E, 0x924949DB, 0x0C06060A, 0x4824246C, 0xB85C5CE4, 
    0x9FC2C25D, 0xBDD3D36E, 0x43ACACEF, 0xC46262A6, 0x399191A8, 0x319595A4, 0xD3E4E437, 0xF279798B, 
    0xD5E7E732, 0x8BC8C843, 0x6E373759, 0xDA6D6DB7, 0x018D8D8C, 0xB1D5D564, 0x9C4E4ED2, 0x49A9A9E0, 
    0xD86C6CB4, 0xAC5656FA, 0xF3F4F407, 0xCFEAEA25, 0xCA6565AF, 0xF47A7A8E, 0x47AEAEE9, 0x10080818, 
    0x6FBABAD5, 0xF0787888, 0x4A25256F, 0x5C2E2E72, 0x381C1C24, 0x57A6A6F1, 0x73B4B4C7, 0x97C6C651, 
    0xCBE8E823, 0xA1DDDD7C, 0xE874749C, 0x3E1F1F21, 0x964B4BDD, 0x61BDBDDC, 0x0D8B8B86, 0x0F8A8A85, 
    0xE0707090, 0x7C3E3E42, 0x71B5B5C4, 0xCC6666AA, 0x904848D8, 0x06030305, 0xF7F6F601, 0x1C0E0E12, 
    0xC26161A3, 0x6A35355F, 0xAE5757F9, 0x69B9B9D0, 0x17868691, 0x99C1C158, 0x3A1D1D27, 0x279E9EB9, 
    0xD9E1E138, 0xEBF8F813, 0x2B9898B3, 0x22111133, 0xD26969BB, 0xA9D9D970, 0x078E8E89, 0x339494A7, 
    0x2D9B9BB6, 0x3C1E1E22, 0x15878792, 0xC9E9E920, 0x87CECE49, 0xAA5555FF, 0x50282878, 0xA5DFDF7A, 
    0x038C8C8F, 0x59A1A1F8, 0x09898980, 0x1A0D0D17, 0x65BFBFDA, 0xD7E6E631, 0x844242C6, 0xD06868B8, 
    0x824141C3, 0x299999B0, 0x5A2D2D77, 0x1E0F0F11, 0x7BB0B0CB, 0xA85454FC, 0x6DBBBBD6, 0x2C16163A };

    return ttable0[plaintext_byte ^ key_candidate];
}


/*
* Kernel that computes the Hamming Weight
*/
__global__ void getHwKernel(int *hw, int *plaintext, int *n)
{
    int idx = blockDim.x*blockIdx.x + threadIdx.x;

    int key_candidate = idx / n[0];

    // AES T-Table LUT
    unsigned int ttable0 [] = 
    { 0xC66363A5, 0xF87C7C84, 0xEE777799, 0xF67B7B8D, 0xFFF2F20D, 0xD66B6BBD, 0xDE6F6FB1, 0x91C5C554, 
    0x60303050, 0x02010103, 0xCE6767A9, 0x562B2B7D, 0xE7FEFE19, 0xB5D7D762, 0x4DABABE6, 0xEC76769A, 
    0x8FCACA45, 0x1F82829D, 0x89C9C940, 0xFA7D7D87, 0xEFFAFA15, 0xB25959EB, 0x8E4747C9, 0xFBF0F00B, 
    0x41ADADEC, 0xB3D4D467, 0x5FA2A2FD, 0x45AFAFEA, 0x239C9CBF, 0x53A4A4F7, 0xE4727296, 0x9BC0C05B, 
    0x75B7B7C2, 0xE1FDFD1C, 0x3D9393AE, 0x4C26266A, 0x6C36365A, 0x7E3F3F41, 0xF5F7F702, 0x83CCCC4F, 
    0x6834345C, 0x51A5A5F4, 0xD1E5E534, 0xF9F1F108, 0xE2717193, 0xABD8D873, 0x62313153, 0x2A15153F, 
    0x0804040C, 0x95C7C752, 0x46232365, 0x9DC3C35E, 0x30181828, 0x379696A1, 0x0A05050F, 0x2F9A9AB5, 
    0x0E070709, 0x24121236, 0x1B80809B, 0xDFE2E23D, 0xCDEBEB26, 0x4E272769, 0x7FB2B2CD, 0xEA75759F, 
    0x1209091B, 0x1D83839E, 0x582C2C74, 0x341A1A2E, 0x361B1B2D, 0xDC6E6EB2, 0xB45A5AEE, 0x5BA0A0FB, 
    0xA45252F6, 0x763B3B4D, 0xB7D6D661, 0x7DB3B3CE, 0x5229297B, 0xDDE3E33E, 0x5E2F2F71, 0x13848497, 
    0xA65353F5, 0xB9D1D168, 0x00000000, 0xC1EDED2C, 0x40202060, 0xE3FCFC1F, 0x79B1B1C8, 0xB65B5BED, 
    0xD46A6ABE, 0x8DCBCB46, 0x67BEBED9, 0x7239394B, 0x944A4ADE, 0x984C4CD4, 0xB05858E8, 0x85CFCF4A, 
    0xBBD0D06B, 0xC5EFEF2A, 0x4FAAAAE5, 0xEDFBFB16, 0x864343C5, 0x9A4D4DD7, 0x66333355, 0x11858594, 
    0x8A4545CF, 0xE9F9F910, 0x04020206, 0xFE7F7F81, 0xA05050F0, 0x783C3C44, 0x259F9FBA, 0x4BA8A8E3, 
    0xA25151F3, 0x5DA3A3FE, 0x804040C0, 0x058F8F8A, 0x3F9292AD, 0x219D9DBC, 0x70383848, 0xF1F5F504, 
    0x63BCBCDF, 0x77B6B6C1, 0xAFDADA75, 0x42212163, 0x20101030, 0xE5FFFF1A, 0xFDF3F30E, 0xBFD2D26D, 
    0x81CDCD4C, 0x180C0C14, 0x26131335, 0xC3ECEC2F, 0xBE5F5FE1, 0x359797A2, 0x884444CC, 0x2E171739, 
    0x93C4C457, 0x55A7A7F2, 0xFC7E7E82, 0x7A3D3D47, 0xC86464AC, 0xBA5D5DE7, 0x3219192B, 0xE6737395, 
    0xC06060A0, 0x19818198, 0x9E4F4FD1, 0xA3DCDC7F, 0x44222266, 0x542A2A7E, 0x3B9090AB, 0x0B888883, 
    0x8C4646CA, 0xC7EEEE29, 0x6BB8B8D3, 0x2814143C, 0xA7DEDE79, 0xBC5E5EE2, 0x160B0B1D, 0xADDBDB76, 
    0xDBE0E03B, 0x64323256, 0x743A3A4E, 0x140A0A1E, 0x924949DB, 0x0C06060A, 0x4824246C, 0xB85C5CE4, 
    0x9FC2C25D, 0xBDD3D36E, 0x43ACACEF, 0xC46262A6, 0x399191A8, 0x319595A4, 0xD3E4E437, 0xF279798B, 
    0xD5E7E732, 0x8BC8C843, 0x6E373759, 0xDA6D6DB7, 0x018D8D8C, 0xB1D5D564, 0x9C4E4ED2, 0x49A9A9E0, 
    0xD86C6CB4, 0xAC5656FA, 0xF3F4F407, 0xCFEAEA25, 0xCA6565AF, 0xF47A7A8E, 0x47AEAEE9, 0x10080818, 
    0x6FBABAD5, 0xF0787888, 0x4A25256F, 0x5C2E2E72, 0x381C1C24, 0x57A6A6F1, 0x73B4B4C7, 0x97C6C651, 
    0xCBE8E823, 0xA1DDDD7C, 0xE874749C, 0x3E1F1F21, 0x964B4BDD, 0x61BDBDDC, 0x0D8B8B86, 0x0F8A8A85, 
    0xE0707090, 0x7C3E3E42, 0x71B5B5C4, 0xCC6666AA, 0x904848D8, 0x06030305, 0xF7F6F601, 0x1C0E0E12, 
    0xC26161A3, 0x6A35355F, 0xAE5757F9, 0x69B9B9D0, 0x17868691, 0x99C1C158, 0x3A1D1D27, 0x279E9EB9, 
    0xD9E1E138, 0xEBF8F813, 0x2B9898B3, 0x22111133, 0xD26969BB, 0xA9D9D970, 0x078E8E89, 0x339494A7, 
    0x2D9B9BB6, 0x3C1E1E22, 0x15878792, 0xC9E9E920, 0x87CECE49, 0xAA5555FF, 0x50282878, 0xA5DFDF7A, 
    0x038C8C8F, 0x59A1A1F8, 0x09898980, 0x1A0D0D17, 0x65BFBFDA, 0xD7E6E631, 0x844242C6, 0xD06868B8, 
    0x824141C3, 0x299999B0, 0x5A2D2D77, 0x1E0F0F11, 0x7BB0B0CB, 0xA85454FC, 0x6DBBBBD6, 0x2C16163A };

    unsigned int b = ttable0[plaintext[idx % n[0]]^key_candidate];

    hw[idx] = 0;
    while (b) 
    {
        hw[idx] += (b & 1);
        b >>= 1;
    }

    /* Loop Unrolling: 
    does not provide any speed advantage compared to while()-loop 
    (because of Compiler optimizations probably)
    hw[idx] = (b & 1); b >>= 1;  //0
    hw[idx] += (b & 1); b >>= 1; //1
    hw[idx] += (b & 1); b >>= 1; //2
    hw[idx] += (b & 1); b >>= 1; //3
    hw[idx] += (b & 1); b >>= 1; //4
    hw[idx] += (b & 1); b >>= 1; //5
    hw[idx] += (b & 1); b >>= 1; //6
    hw[idx] += (b & 1); b >>= 1; //7
    hw[idx] += (b & 1); b >>= 1; //8
    hw[idx] += (b & 1); b >>= 1; //9
    hw[idx] += (b & 1); b >>= 1; //10
    hw[idx] += (b & 1); b >>= 1; //11
    hw[idx] += (b & 1); b >>= 1; //12
    hw[idx] += (b & 1); b >>= 1; //13
    hw[idx] += (b & 1); b >>= 1; //14
    hw[idx] += (b & 1); b >>= 1; //15
    hw[idx] += (b & 1); b >>= 1; //16
    hw[idx] += (b & 1); b >>= 1; //17
    hw[idx] += (b & 1); b >>= 1; //18
    hw[idx] += (b & 1); b >>= 1; //19
    hw[idx] += (b & 1); b >>= 1; //20
    hw[idx] += (b & 1); b >>= 1; //21
    hw[idx] += (b & 1); b >>= 1; //22
    hw[idx] += (b & 1); b >>= 1; //23
    hw[idx] += (b & 1); b >>= 1; //24
    hw[idx] += (b & 1); b >>= 1; //25
    hw[idx] += (b & 1); b >>= 1; //26
    hw[idx] += (b & 1); b >>= 1; //27
    hw[idx] += (b & 1); b >>= 1; //28
    hw[idx] += (b & 1); b >>= 1; //29
    hw[idx] += (b & 1); b >>= 1; //30
    hw[idx] += (b & 1); b >>= 1; //31
    */
}

/*
*	First kernel for computation of the Pearson Correlation Coefficient
*/
__global__ void CorrCoefKernel_Naiv(double *result, int *x, int *y, int first_col) //row = NUMBER_OF_TRACES, col = POINTS_PER_TRACE
{
    int i = threadIdx.x + first_col;

    _Uint32t sum_x  = 0;
    _Uint32t sum_y  = 0;

    for(int j = 0; j < NUMBER_OF_TRACES; j++)
    {
        sum_x  += x[i+j*POINTS_PER_TRACE];
        sum_y  += y[j];
    }

    double x_average = sum_x/NUMBER_OF_TRACES;
    double y_average = sum_y/NUMBER_OF_TRACES;

    double dividend = 0;
    double divisor1 = 0;      
    double divisor2 = 0;

    for(int j = 0; j < NUMBER_OF_TRACES; j++)
    {
        dividend += (x[i+j*POINTS_PER_TRACE] - x_average)*(y[j] - y_average); 
        divisor1 += (x[i+j*POINTS_PER_TRACE] - x_average)*(x[i+j*POINTS_PER_TRACE] - x_average);  
        divisor2 += (y[j] - y_average)*(y[j] - y_average); 
    }

    double divisor = sqrt(divisor1)*sqrt(divisor2);

    if ((dividend == 0) || (divisor == 0))
    {
        result[i-first_col] = 0.0;
    }else{
        result[i-first_col] = dividend/divisor;
    }		
}

/*
*	Second version of the kernel for computation of the Pearson Correlation Coefficient using shared memory
*/
__global__ void CorrCoefKernel_SharedMem(double *result, int *x, int *y, int first_col)
{
    int i = threadIdx.x + first_col;

    __shared__ int Xs[NUMBER_OF_TRACES];
    // __shared__ int Ys[NUMBER_OF_TRACES];

    _Uint32t sum_x  = 0;
    _Uint32t sum_y  = 0;


    for(int j = 0; j < NUMBER_OF_TRACES; j++)
    {
        Xs[j] = x[i+j*POINTS_PER_TRACE];
        // Ys[j] = y[j];
        sum_x  += Xs[j];
        sum_y  += y[j];
    }

    double x_average = sum_x/NUMBER_OF_TRACES;
    double y_average = sum_y/NUMBER_OF_TRACES;

    double dividend = 0;
    double divisor1 = 0;      
    double divisor2 = 0;

    for(int j = 0; j < NUMBER_OF_TRACES; j++)
    {
        dividend += (Xs[j] - x_average)*(y[j] - y_average); 
        divisor1 += (Xs[j] - x_average)*(Xs[j] - x_average);  
        divisor2 += (y[j] - y_average)*(y[j] - y_average); 
    }

    double divisor = sqrt(divisor1)*sqrt(divisor2);

    if ((dividend == 0) || (divisor == 0))
    {
        result[i-first_col] = 0.0;
    }else{
        result[i-first_col] = dividend/divisor;
    }		
}

/*
* Function to calculate the Pearson Correlation Coefficient
*/
double get_Corr_Coef(int *x, int *y, int n)
{
    _Uint32t sum_x  = 0;
    _Uint32t sum_y  = 0;

    for(int i = 0; i < n; i++)
    {
        sum_x  += x[i];
        sum_y  += y[i];
    }

    long double x_average = sum_x/n;
    long double y_average = sum_y/n;

    long double dividend = 0;
    long double divisor1 = 0;
    long double divisor2 = 0;

    for(int i = 0; i < n; i++)
    {
        dividend += (x[i] - x_average)*(y[i] - y_average); 
        divisor1 += (x[i] - x_average)*(x[i] - x_average); 
        divisor2 += (y[i] - y_average)*(y[i] - y_average); 
    }

    long double divisor = sqrt(divisor1)*sqrt(divisor2);

    if ((dividend == 0) || (divisor == 0))
    {
        return 0.0;
    }else{
        return dividend/divisor;
    }		
}

// Helper function for using CUDA to calculate Hamming Weight in parallel.
hipError_t getHwWithCuda(int *hw, int *plaintext)
{
    int *dev_plaintext = 0;
    int *dev_n = 0;
    int *dev_hw = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Start measuring time
    //const clock_t begin_time_mal = clock();

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_hw, NUMBER_OF_TRACES * NUMBER_OF_KEY_CANDIDATES * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_plaintext, NUMBER_OF_TRACES * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_n, NUMBER_OF_TRACES * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_plaintext, plaintext, NUMBER_OF_TRACES * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_n, &NUMBER_OF_TRACES_INT, NUMBER_OF_TRACES * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    getHwKernel<<<2500, 1024>>>(dev_hw, dev_plaintext, dev_n);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "getHwKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching getHwKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(hw, dev_hw, NUMBER_OF_TRACES * NUMBER_OF_KEY_CANDIDATES * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_hw);
    hipFree(dev_plaintext);

    return cudaStatus;
}

// Helper function for computation of the correlation coefficient using CUDA.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
hipError_t computeCoeffWithCuda(double *cc, int *traces, int *hw)
{
    int *dev_traces = 0;
    int *dev_hw = 0;
    double *dev_cc = 0;
    hipError_t cudaStatus;
    //size_t pitch;

    /*dev_traces = traces;*/

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_cc, (TRACE_ENDPOINT - TRACE_STARTPOINT) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_hw, NUMBER_OF_TRACES * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_traces, NUMBER_OF_TRACES * POINTS_PER_TRACE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    /*cudaStatus = hipMallocPitch(&dev_traces, &pitch,
    POINTS_PER_TRACE * sizeof(int), NUMBER_OF_TRACES);
    if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMallocPitch failed!");
    goto Error;
    }*/


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_hw, hw, NUMBER_OF_TRACES * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_traces, traces, NUMBER_OF_TRACES * POINTS_PER_TRACE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    /*cudaStatus = hipMemcpy2D(dev_traces, pitch, &traces_array, n * sizeof(int), n * sizeof(int), m, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy2D failed!");
    goto Error;
    }*/


    // Launch a kernel on the GPU with one thread for each tracepoint.
#ifdef NAIVE
    CorrCoefKernel_Naiv<<<1, TRACE_ENDPOINT-TRACE_STARTPOINT>>>(dev_cc, dev_traces, dev_hw, TRACE_STARTPOINT);
#endif
#ifdef SHARED
    CorrCoefKernel_SharedMem<<<1, TRACE_ENDPOINT-TRACE_STARTPOINT>>>(dev_cc, dev_traces, dev_hw, TRACE_STARTPOINT);
#endif
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CorrCoefKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching CorrCoefKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(cc, dev_cc, (TRACE_ENDPOINT - TRACE_STARTPOINT) * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_cc);
    hipFree(dev_hw);

    //for (int p = 0; p < POINTS_PER_TRACE; p++)
    //{
    //   hipFree(dev_traces[p]);
    //}
    hipFree(dev_traces);


    return cudaStatus;
}



int main()
{
    // #################### OWN PROGRAM #####################

    //printDevInfos();

    // Start measuring time
    const clock_t begin_time = clock();

    // Initialize trace array
    //array2d<int> traces_array(POINTS_PER_TRACE,NUMBER_OF_TRACES);

    int *traces;
    traces = new int [POINTS_PER_TRACE*NUMBER_OF_TRACES];
    //for (int i = 0; i < POINTS_PER_TRACE; i++)
    //{
    //	traces[i] = new int[NUMBER_OF_TRACES];
    //}

    //// Read traces and store in array
    read_traces(traces, TRACE_FILE);
    //read_traces_array(traces_array, TRACE_FILE);

    //int *dev_traces = 0;
    //hipError_t cudaStatus;

    //    cudaStatus = hipMalloc((void**)&dev_traces, NUMBER_OF_TRACES * POINTS_PER_TRACE * sizeof(int));
    //   if (cudaStatus != hipSuccess) {
    //       fprintf(stderr, "hipMalloc failed!");
    //       return 1;
    //   }

    //    cudaStatus = hipMemcpy(dev_traces, traces, NUMBER_OF_TRACES * POINTS_PER_TRACE * sizeof(int), hipMemcpyHostToDevice);
    //   if (cudaStatus != hipSuccess) {
    //       fprintf(stderr, "hipMemcpy failed!");
    //       return 1;
    //   }

    // Stop measuring time
    std::cout << float( clock () - begin_time ) /  CLOCKS_PER_SEC << "sec" << endl;


    // Start measuring time
    const clock_t begin_time_plaintexts = clock();

    // Initialize plaintext array
    unsigned _int8 **plaintexts;
    plaintexts = new unsigned _int8 *[NUMBER_OF_TEXTS];
    for (int i = 0; i < NUMBER_OF_TEXTS; i++)
    {
        plaintexts[i] = new unsigned _int8[BYTES_PER_TEXT];
    }

    // Read plaintexts and store in array
    read_texts(plaintexts, PLAINTEXT_FILE);

    // Stop measuring time
    std::cout << float( clock () - begin_time_plaintexts ) /  CLOCKS_PER_SEC << "sec" << endl;


    // Start measuring time
    const clock_t begin_time_calculation = clock();

#ifdef HW_SL
    int *hw_sl;
    hw_sl = new int [NUMBER_OF_TRACES];
#endif

    int *key;
    key = new int [BYTES_PER_KEY];

    // Initialize corr array (event. doch nicht n�tig)
    double **corr;
    corr = new double *[256];
    for (int i = 0; i < 256; i++)
    {
        corr[i] = new double[TRACE_ENDPOINT - TRACE_STARTPOINT];
    }

    // Loop through all key bytes
    for (int key_byte = 0; key_byte < BYTES_PER_KEY; key_byte++)
    {
        cout << "Compute key byte " << key_byte << " ..." << endl;

        double highest_cc = -1.0;
        double cc = -1.0;

        // ################# HW PARALLEL ######################
#ifdef HW_PL
        // Start measuring time
        const clock_t begin_time_hw_pl = clock();

        int *hw_pl;
        hw_pl = new int [NUMBER_OF_TRACES * NUMBER_OF_KEY_CANDIDATES];

        int* plaintext_at_key_byte;
        plaintext_at_key_byte = new int[NUMBER_OF_TRACES];

        for(int i = 0; i < NUMBER_OF_TRACES; i++)
        {
            plaintext_at_key_byte[i] = plaintexts[i][key_byte];
        }

        // Calculate Hamming Weight in parallel.
        hipError_t cudaStatusHw = getHwWithCuda(hw_pl, plaintext_at_key_byte);
        if (cudaStatusHw != hipSuccess) {
            fprintf(stderr, "getHwWithCuda failed!");
            return 1;
        }

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        cudaStatusHw = hipDeviceReset();
        if (cudaStatusHw != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }

        // Stop measuring time
        std::cout << " HW parallel:  " << float( clock () - begin_time_hw_pl ) /  CLOCKS_PER_SEC << "sec" << endl;
#endif


        // ##################### HW SERIELL #####################

        // Start measuring time
        const clock_t begin_time_hw_sl = clock();

        // Loop through all key candidates
        for (int key_candidate = 0; key_candidate < NUMBER_OF_KEY_CANDIDATES; key_candidate++)
        {   

            // Start measuring time
            const clock_t begin_time_hw2 = clock();
#ifdef HW_SL
            // Measure hamming weight for every trace
            for (int trace = 0; trace < NUMBER_OF_TRACES; trace++)
            {
                // Calculate the hamming weight
                hw_sl[trace] = get_Hw(get_TTable_Out(plaintexts[trace][key_byte], key_candidate));

                // Compares HW values of parallel and seriell computation
                /*if(hw_sl[trace] != hw_pl[key_candidate*NUMBER_OF_TRACES + trace])
                std::cout << "Error (HW)!! " << hw_sl[trace] << " vs " << hw_pl[key_candidate*NUMBER_OF_TRACES + trace] << endl;*/
            }
#endif
            // +++ PM End +++

            // Stop measuring time
            //std::cout << "HW seriell:       " << float( clock () - begin_time_hw2 ) /  CLOCKS_PER_SEC << "sec" << endl;


            // Calculate Correlation Coefficient 
#ifdef GPU
#ifdef HW_SL
            hipError_t cudaStatus = computeCoeffWithCuda(corr[key_candidate], traces, hw_sl);
#else ifdef HW_PL
            hipError_t cudaStatus = computeCoeffWithCuda(corr[key_candidate], traces, hw_pl);
#endif
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "computeCoeffWithCuda failed!");
                return 1;
            }
#endif
            for (int trace_point = TRACE_STARTPOINT; trace_point < TRACE_ENDPOINT; trace_point++)
            {
#ifdef CPU
                // Create "Slice" of Traces at certain point
                int *traces_at_trace_point;
                traces_at_trace_point = new int [NUMBER_OF_TRACES];

                for (int t = 0; t < NUMBER_OF_TRACES; t++)
                {
                    traces_at_trace_point[t] = traces[trace_point+t*NUMBER_OF_TRACES];
                }

                // Correlation Coefficient 
#ifdef HW_SL
                cc = get_Corr_Coef(traces_at_trace_point, hw_sl, NUMBER_OF_TRACES);
#else ifdef HW_PL
                cc = get_Corr_Coef(traces_at_trace_point, hw_pl, NUMBER_OF_TRACES);
#endif

                delete[] traces_at_trace_point;
#endif
#ifdef GPU
                cc = corr[key_candidate][trace_point-TRACE_STARTPOINT];
#endif
                if(cc > highest_cc)
                {
                    highest_cc = cc;
                    key[key_byte] = key_candidate;
                    //highest_trace_point = trace_point;
                    /*cout << "Highest CC = " << highest_cc << ", Key Candidate = " << key_candidate << endl;*/
                }
            }
        }
        highest_cc = -1.0;

#ifdef HW_PL
        delete[] hw_pl;
#endif HW_PL
        // Stop measuring time
        /*std::cout << " HW seriell:   " << float( clock () - begin_time_hw_sl ) /  CLOCKS_PER_SEC << "sec" << endl;*/
    } 

    cout << "CIPHER KEY =";
    for(int i = 0; i < BYTES_PER_KEY; i++)
    {
        cout << hex << " " << key[i];
    }
    cout << endl;


    // deleting everything
    std::cout << "GPU not reseted yet, Time: " << float( clock () - begin_time_calculation ) /  CLOCKS_PER_SEC << "sec" << endl;

    /*for (int i = 0; i < NUMBER_OF_TRACES; i++)
    {
    delete[] traces[i];
    }*/
    for (int i = 0; i < NUMBER_OF_TEXTS; i++)
    {
        delete[] plaintexts[i];
    }
    /*delete[] traces;*/
    //traces_array.~array2d();
    delete[] traces;
    delete[] plaintexts;
    delete[] key;

#ifdef hw_sl
    delete[] hw_sl;
#endif

    for (int i = 0; i < 256; i++)
    {
        delete[] corr[i];
    }
    delete[] corr;

    /* hipFree(dev_traces);*/

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!");
    return 1;*/

    // 	CORRECT CIPHER KEY: 2b  7e  15  16  28  ae  d2  a6  ab  f7  15  88  09  cf  4f  3c
    //                 dec: 043 126 021 022 040 174 210 166 171 247 021 136 009 207 079 060		 

    // Stop measuring time
    std::cout << float( clock () - begin_time_calculation ) /  CLOCKS_PER_SEC << "sec" << endl;

    return 0;
}
